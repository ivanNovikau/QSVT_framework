#include "hip/hip_runtime.h"
#include "../third_party/hip/hip_runtime_api.h"

#include <sstream>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <stdexcept>
#include <filesystem>
#include <math.h>
#include <vector>
#include <map>
#include <iterator>
#include <algorithm>
#include <time.h>
#include <limits>
#include <chrono>
#include <stdarg.h>
#include <memory>
#include <numeric>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include "H5Cpp.h"

using namespace std;

struct FUNC_
{
    string sel;
    int32_t parity; // 0 - even, 1 - odd;
};

const vector<FUNC_> avail_functions_ = {
   {"inversion", 1}
};

struct FUNC_DATA_
{
    int32_t id;
    int32_t parity;
    double param;
    double coef_norm;
};
__constant__ FUNC_DATA_ function_d_;

void calculate_coefficients(
    const uint32_t& Nd, const uint32_t& N_coefs_avail, const FUNC_DATA_& function_h,
    double*& coefs_real, double*& coefs_imag, uint32_t& N_coefs
);
void construct_polynomial(
    const double* coefs_real, const uint32_t& N_coefs,
    const FUNC_DATA_& function_h, const uint32_t& Nx_half, 
    double*& x, double*& pol, double*& orig_func, double& err_res
);
void save_coefs(
    const double& param, 
    const double& err_res, 
    const uint32_t& N_coefs, 
    double* coefs_real, 
    double* coefs_imag,
    const double& coef_norm,
    const uint32_t& Nx_half, 
    const double* x, const double* pol, const double* orig_func
);

__global__ 
void calc_coefs_odd(
    uint32_t Nd, uint32_t N_coefs_device, double *coefs_real, double *coefs_imag
);

bool compare_strings(const string& line1, const string& line2);
void get_current_date_time(string& line_date_time);



int main(int argc, char *argv[])
{
    int nDevices;
    uint32_t Nd;   // initial number of coefficients in the polynomial;
    double param;  // function main parameter: kappa, time etc.
    string sel_function; // ID of the function to approximate;
    FUNC_DATA_ function_h;
    uint32_t id_arg;

    cout << "--- Fourier approach ---" << endl;

    // --- INPUT parameters ---
    if(argc < 7)
    {
        cout << "Error: some input parameters are missing." << endl;
        return -1;
    }

    id_arg = 1;
    while(id_arg < (argc - 1))
    {
        if(compare_strings(argv[id_arg], "-sel_function"))
        {
            id_arg += 1;
            sel_function = string (argv[id_arg]);
        }
        if(compare_strings(argv[id_arg], "-param"))
        {
            id_arg += 1;
            param = stod(string (argv[id_arg]));
        }
        if(compare_strings(argv[id_arg], "-Nd"))
        {
            id_arg += 1;
            Nd = stoi(string (argv[id_arg]));
        }
        ++id_arg;
    }

    function_h.id = -1;
    for(int32_t ii = 0; ii < avail_functions_.size(); ii++)
        if(compare_strings(avail_functions_[ii].sel, sel_function))
            function_h.id = ii;
    if(function_h.id == -1)
    {
        cout << "Error: the function with ID = {" << sel_function << "} is not defined." << endl;
        return -1;
    }
    function_h.parity = avail_functions_[function_h.id].parity;
    function_h.param = param;
    function_h.coef_norm = 0.125;

    hipGetDeviceCount(&nDevices);
    if(nDevices == 0)
    {
        cout << "Error: GPU devices that support CUDA are not found." << endl;
        return -1;
    }

    cout << "Function to approximate: " << avail_functions_[function_h.id].sel << "\n";
    cout << "Its parity: \t" << function_h.parity << "\n";
    cout << "Parameter: \t" << param << "\n";
    cout << "Nd: \t" << Nd << "\n";
    cout << "N of avail. GPU devices: " << nDevices << endl;

    // assume that all devices have the same properties:
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double coef_mem = 0.8;
    uint32_t N_coefs_avail = coef_mem * prop.totalGlobalMem / (2*sizeof(double));
    cout << "Avail. GPU mem for coefs. (MB): " << 
        coef_mem * prop.totalGlobalMem / (1024 * 1024.) << "\n";

    for(int id_device = 0; id_device < nDevices; id_device++)
    {
        hipSetDevice(id_device);
        hipMemcpyToSymbol(HIP_SYMBOL(function_d_), &function_h, sizeof(FUNC_DATA_));
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
    }

    // calculate the polynomial coefficients:
    double* coefs_real;
    double* coefs_imag;
    uint32_t N_coefs;
    calculate_coefficients(Nd, N_coefs_avail, function_h, coefs_real, coefs_imag, N_coefs);
    
    // construct the resulting polynomial and estimate the error:
    uint32_t Nx_half = 2001; 
    double *x, *pol, *orig_fun;
    double err_res;
    construct_polynomial(coefs_real, N_coefs, function_h, Nx_half, x, pol, orig_fun, err_res);
  
    // --- Save the coefficients to the .hdf5 file ---
    save_coefs(param, err_res, N_coefs, coefs_real, coefs_imag, function_h.coef_norm, Nx_half, x, pol, orig_fun);
    

    return 0;
}


__device__ __forceinline__
double F_CALC(const double& x) // if modified, change also F_CALC_HOST.
{
    // inversion function:
    if(function_d_.id == 0)
    {
        double kappa = function_d_.param;
        return (function_d_.coef_norm/kappa) * (1. - exp(-pow(5*kappa*x, 2))) / x;
    }
    return 0; // function is missing;
}

double F_CALC_HOST(const double& x, const FUNC_DATA_& function_h) // if modified, change also F_CALC.
{
    // inversion function:
    if(function_h.id == 0)
    {
        double kappa = function_h.param;
        return (function_h.coef_norm/kappa) * (1. - exp(-pow(5*kappa*x, 2))) / x;
    }
    return 0; // function is missing;
}


void calculate_coefficients(
    const uint32_t& Nd, const uint32_t& N_coefs_avail, const FUNC_DATA_& function_h,
    double*& coefs_real, double*& coefs_imag, uint32_t& N_coefs
){
    N_coefs = int(Nd/2.);

    // ATTENTION: the case whether the number of coefficients > number of coefficients that can be stored
    //  on a single GPU is not debugged:
    uint32_t N_coefs_device = (N_coefs > N_coefs_avail) ? N_coefs_avail: N_coefs;
    uint32_t N_iter         = int(N_coefs/N_coefs_avail) + 1;
    double* coefs_real_d;
    double* coefs_imag_d;

    // ATTENTION: N_coefs maybe too large, in this case it will be necessary 
    // to create new arrays coefs_real and coefs_imag at each iteration:
    coefs_real = new double[N_coefs]; 
    coefs_imag = new double[N_coefs];

    cout << "\n---\n";
    cout << "Required mem. for coefs. (MB): " << 
        2*sizeof(double) * N_coefs / (1024 * 1024.) << endl;

    hipMalloc((void**) &(coefs_real_d), N_coefs_device * sizeof(double));
    hipMalloc((void**) &(coefs_imag_d), N_coefs_device * sizeof(double));
    uint32_t N_coefs_device_init = N_coefs_device;
    for(auto counter_iter = 0; counter_iter < N_iter; counter_iter++)
    {
        if(counter_iter == N_iter - 1)
            N_coefs_device = N_coefs - N_coefs_device * counter_iter;

        hipMemset(coefs_real_d, 0.0, N_coefs_device * sizeof(double));
        hipMemset(coefs_imag_d, 0.0, N_coefs_device * sizeof(double));

        uint32_t N_threads_per_block = (N_coefs_device < 1024)?  N_coefs_device: 1024;
        uint32_t N_cuda_block = 
            (N_coefs_device <= N_threads_per_block) ? 
                1:
                int(N_coefs_device / N_threads_per_block) + 1;
        
        cout << "\n\t *** Iteration: " << counter_iter        << " ***\n";
        cout << "\tN-coefs per device: " << N_coefs_device << "\n";
        cout << "\tN-blocks: "  << N_cuda_block        << "\n";
        cout << "\tN-threads: " << N_threads_per_block << "\n";

        // CALCULATE the polynomial coefficients for the odd function:
        if(function_h.parity == 1)
            calc_coefs_odd<<<N_cuda_block, N_threads_per_block>>>(
                Nd, N_coefs_device, coefs_real_d, coefs_imag_d
            );

        hipMemcpy(
            coefs_real + N_coefs_device_init * counter_iter, 
            coefs_real_d,
            N_coefs_device * sizeof(double),
            hipMemcpyDeviceToHost
        );
        hipMemcpy(
            coefs_imag + N_coefs_device_init * counter_iter, 
            coefs_imag_d,
            N_coefs_device * sizeof(double),
            hipMemcpyDeviceToHost
        );

        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
    }
}


void construct_polynomial(
    const double* coefs_real, const uint32_t& N_coefs,
    const FUNC_DATA_& function_h, const uint32_t& Nx_half, 
    double*& x, double*& pol, double*& orig_func, double& err_res
){
    // ATTENTION: here we assume that all coefficients can be saved on a single GPU.

    cout << "\nConstruction of the polynomial...\n";
    x         = new double[2*Nx_half];
    pol       = new double[2*Nx_half];
    orig_func = new double[2*Nx_half];

    // x-grid:
    double inv_param = 1./function_h.param;
    double dx = (1 - inv_param) / (Nx_half - 1);
    for(auto ii = 0; ii < Nx_half; ii++)
        x[ii] = -1 + dx * ii;
    for(auto ii = 0; ii < Nx_half; ii++)
        x[Nx_half + ii] = inv_param + dx * ii;

    // Construct the polynomial (using only real coefficients) for the ODD function:
    if(function_h.parity == 1)
        for(auto id_x = 0; id_x < (2*Nx_half); id_x++)
        {
            double pol_one = 0;
            double x_one = x[id_x];
            for(auto id_coef = 1; id_coef < N_coefs+1; id_coef++)
                pol_one += coefs_real[id_coef - 1] * cos((2*id_coef-1)*acos(x_one));
            pol[id_x] = pol_one;
        }

    // Construct the original function:
    for(auto id_x = 0; id_x < (2*Nx_half); id_x++)
        orig_func[id_x] = F_CALC_HOST(x[id_x], function_h);

    // Estimate the absolute error of the polynomial:
    err_res = 0;
    for(auto id_x = 0; id_x < (2*Nx_half); id_x++)
    {
        double err1 = abs(pol[id_x] - orig_func[id_x]);
        if(err1 > err_res)
            err_res = err1;
    }
    cout << "Resulting approximation error: " << std::scientific << setprecision(3) << err_res << endl;
}


__global__ void calc_coefs_odd(
    uint32_t Nd, uint32_t N_coefs_device, double *coefs_real, double *coefs_imag 
){
    // printf("id-thread = %u, parity = %d\n", threadIdx.x, function_d_.parity);
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t N_quad = 2 * Nd; // or try 1, 4, 8, 16

    if(idx < N_coefs_device)
    {
        uint32_t ii = 2 * idx + 1;
        double coef_in_front = pow(-1, ii) / N_quad;
        double sum_temp_real = 0;
        double sum_temp_imag = 0;
        double temp;
        double th;
        for(uint32_t kk = 0; kk < 2*N_quad; kk++)
        {
            th = M_PI * kk / N_quad;
            temp = F_CALC(-cos(th));
            sum_temp_real += temp * cos(ii * th);
            sum_temp_imag += temp * sin(ii * th);
        }
        coefs_real[idx] = coef_in_front * sum_temp_real;
        coefs_imag[idx] = coef_in_front * sum_temp_imag;
    }
}


void save_coefs(
    const double& param, 
    const double& err_res, 
    const uint32_t& N_coefs, 
    double* coefs_real, 
    double* coefs_imag,
    const double& coef_norm,
    const uint32_t& Nx_half, 
    const double* x, const double* pol, const double* orig_func
){
    std::stringstream sstr;
    sstr << "./coef_xodd_" << int(param) << "_" << round(-log10(err_res)) << ".hdf5";

    string filename_hdf5 = sstr.str(); 
    H5::H5File* f_ = new H5::H5File(filename_hdf5, H5F_ACC_TRUNC);
    H5::Group grp_basic(f_->createGroup("basic"));
    H5::Group grp_coefs(f_->createGroup("coefs"));
    H5::Group grp_functions(f_->createGroup("functions"));

    // description of the data:
    string descr = "coefs. for the inversion function";
    H5::StrType dtype_descr(H5::PredType::C_S1, descr.size()+1);
    H5::DataSet dataset_descr = grp_basic.createDataSet(
        "descr", 
        dtype_descr, 
        H5::DataSpace(H5S_SCALAR)
    );
    dataset_descr.write(descr, dtype_descr);

    // save the date of the simulation:
    string str_date_time;
    get_current_date_time(str_date_time);

    H5::StrType dtype_str_time(H5::PredType::C_S1, str_date_time.size()+1);
    H5::DataSet dataset_str_time = grp_basic.createDataSet(
        "date-of-simulation", 
        dtype_str_time, 
        H5::DataSpace(H5S_SCALAR)
    );
    dataset_str_time.write(str_date_time, dtype_str_time);

    // save the function parameter:
    H5::DataSet dataset_param = grp_basic.createDataSet(
        "param", 
        H5::PredType::NATIVE_DOUBLE, 
        H5::DataSpace(H5S_SCALAR)
    );
    dataset_param.write((int*) &param, H5::PredType::NATIVE_DOUBLE);

    // save the approximation error:
    H5::DataSet dataset_err = grp_basic.createDataSet(
        "eps", 
        H5::PredType::NATIVE_DOUBLE, 
        H5::DataSpace(H5S_SCALAR)
    );
    dataset_err.write((int*) &err_res, H5::PredType::NATIVE_DOUBLE);

    // save the function normalization factor:
    H5::DataSet dataset_factor_norm = grp_basic.createDataSet(
        "coef_norm", 
        H5::PredType::NATIVE_DOUBLE, 
        H5::DataSpace(H5S_SCALAR)
    );
    dataset_factor_norm.write((int*) &coef_norm, H5::PredType::NATIVE_DOUBLE);

    // save the coefficients:
    hsize_t dims_coefs[] = {N_coefs};
    H5::DataSpace dspace_coefs(1, dims_coefs);
    H5::DataSet dataset_real = grp_coefs.createDataSet(
        "real", 
        H5::PredType::NATIVE_DOUBLE, 
        dspace_coefs
    );
    dataset_real.write(coefs_real, H5::PredType::NATIVE_DOUBLE);

    H5::DataSet dataset_imag = grp_coefs.createDataSet(
        "imag", 
        H5::PredType::NATIVE_DOUBLE, 
        dspace_coefs
    );
    dataset_imag.write(coefs_imag, H5::PredType::NATIVE_DOUBLE);

    // save the x-grid:
    hsize_t dims_x[] = {2*Nx_half};
    H5::DataSpace dspace_x(1, dims_x);
    H5::DataSet dataset_x = grp_functions.createDataSet(
        "x", 
        H5::PredType::NATIVE_DOUBLE, 
        dspace_x
    );
    dataset_x.write(x, H5::PredType::NATIVE_DOUBLE);

    // save the original function:
    hsize_t dims_orig[] = {2*Nx_half};
    H5::DataSpace dspace_orig(1, dims_orig);
    H5::DataSet dataset_orig = grp_functions.createDataSet(
        "orig", 
        H5::PredType::NATIVE_DOUBLE, 
        dspace_orig
    );
    dataset_orig.write(orig_func, H5::PredType::NATIVE_DOUBLE);

    // save the constructed polynomial:
    hsize_t dims_pol[] = {2*Nx_half};
    H5::DataSpace dspace_pol(1, dims_pol);
    H5::DataSet dataset_pol = grp_functions.createDataSet(
        "pol", 
        H5::PredType::NATIVE_DOUBLE, 
        dspace_pol
    );
    dataset_pol.write(pol, H5::PredType::NATIVE_DOUBLE);

    delete f_;

    cout << "Data are saved to the file: " << filename_hdf5 << endl;
}


bool compare_strings(const string& line1, const string& line2)
{
    string new_line1(line1), new_line2(line2);
    transform(new_line1.begin(), new_line1.end(), new_line1.begin(), ::tolower);
    transform(new_line2.begin(), new_line2.end(), new_line2.begin(), ::tolower);

    if(new_line1.compare(new_line2) == 0)
        return true;
    else
        return false;
}

void get_current_date_time(string& line_date_time)
{
    time_t rawtime;
    struct tm * timeinfo;
    char buffer[80];

    time (&rawtime);
    timeinfo = localtime(&rawtime);
    strftime(buffer, sizeof(buffer), "%m-%d-%Y %H:%M:%S", timeinfo);
    line_date_time = string(buffer);
}







